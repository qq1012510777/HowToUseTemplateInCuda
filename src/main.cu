#include "hip/hip_runtime.h"
#include "Triangle_r.cuh"

template <typename T>
__global__ void TestKernel(cuDFNsys::Triangle_r<T> *K, uint count)
{
    //printf("kernel\n");
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    //printf("thread: %d\n", i);
    if (i > count - 1)
        return;

    vec2_select_t<T> KJ;
    KJ.x = 1000;
    KJ.y = 10000;

    K[i].Coordinate[0] = KJ;
    //printf("%f,, %f\n", K[i].Coordinate[0].x, K[i].Coordinate[0].y);
};
template __global__ void TestKernel(cuDFNsys::Triangle_r<double> *K, uint count);
template __global__ void TestKernel(cuDFNsys::Triangle_r<float> *K, uint count);

int main()
{
    using data_type_ = double;
    int dev = 0;
    hipSetDevice(dev);

    vec2_select_t<data_type_> HT[3];
    HT[0].x = 1, HT[0].y = 2;
    HT[1].x = 3, HT[1].y = 8;
    HT[2].x = 7, HT[2].y = 4;

    cuDFNsys::Triangle_r<data_type_> HS;

    cuDFNsys::Triangle_r<data_type_> HH(HT[0], HT[1], HT[2], 1.0);

    cout << "First node/end of the triangle: " << HH.Coordinate[0].x << ", " << HH.Coordinate[0].y << endl;
    cout << "Tag: " << HH.Tag << endl;

    int NodeNO = 1;
    vec2_select_t<data_type_> JK = HH.GetOneEnd(NodeNO);
    cout << "Node " << (NodeNO > 2 ? 2 : NodeNO) << ": " << JK.x << ", " << JK.y << endl;

    //---test kernel--------------

    thrust::host_vector<cuDFNsys::Triangle_r<data_type_>> hostV(1);
    hostV[0] = HH;
    //cout << "host_vec: " << hostV[0].Coordinate[0].x << ", " << hostV[0].Coordinate[0].y << endl;

    thrust::device_vector<cuDFNsys::Triangle_r<data_type_>> DevV(1);
    DevV = hostV;
    // cout << "dev_vec: " << DevV[0].Coordinate[0].x << ", " << DevV[0].Coordinate[0].y << endl;
    // do not uncomment the above sentence, it is stupid

    cuDFNsys::Triangle_r<data_type_> *PntD = thrust::raw_pointer_cast(DevV.data());
    // cout << "devPnter_vec: " << PntD[0].Coordinate[0].x << ", " << PntD[0].Coordinate[0].y << endl;
    // do not uncomment the above sentence, it is stupid

    TestKernel<<<1, 32>>>(PntD, 1);
    hipDeviceSynchronize();
    hostV = DevV;
    HH = hostV[0];

    cout << "After kernel, the first node/end of the triangle: " << HH.Coordinate[0].x << ", " << HH.Coordinate[0].y << endl;
    return 0;
};
